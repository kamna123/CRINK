#define _NTHREAD 512
#define _NBLOCK 65535
#include<hip/hip_runtime.h>

__global__ void _AFFINE_KERNEL(int* ,int ,int ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
int main()
{
    int a[50],i;
    for(i=0;i<50;i++)
    {
        a[i]=2*i;
    }
	int _SZ_a_1 = 50;
	int *_DEV_a;
	hipMalloc((void**) &_DEV_a, sizeof(int)*_SZ_a_1);
	hipMemcpy(_DEV_a, a, sizeof(int)*_SZ_a_1, hipMemcpyHostToDevice);
	float _NUM_THREADS = 50,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=_NUM_THREADS;
	}
	else {
		 _THREADS.x=_NTHREAD;
		_NUM_BLOCKS=_NUM_THREADS/512;
		if(_NUM_BLOCKS<_NBLOCK)
			_BLOCKS.x=_NUM_BLOCKS;
		else {
			_BLOCKS.x=_NBLOCK;
			int temp=_NUM_BLOCKS;
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
		}
	}
	int _CUDA_TILE;
	for(i=0;i<46;i+=8)
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_a, _SZ_a_1, 1, i, 0, 46, _CUDA_TILE);
	hipDeviceSynchronize();
	hipMemcpy(a, _DEV_a, sizeof(int)*_SZ_a_1, hipMemcpyDeviceToHost);
    return 0;
}



__global__ void _AFFINE_KERNEL(int* a,int _SZ_a_1,int phi_count, int CUDA_i, int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_i<=i)&&(i<(CUDA_i+8))&&(i<CUDA_U_i)){
		a[i+10]=a[i+2];
}}

