#define _NTHREAD 512
#define _NBLOCK 65535
#include<hip/hip_runtime.h>

__global__ void _AFFINE_KERNEL(int* ,int ,int ,int* ,int ,int ,int ,int ,int ,int ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
int main()
{
    int x[20][20];
    int y[20][20];
    int a[20],i,j,k;
    int n=20;
    for(i=0;i<20;i++)
    for(j=0;j<20;j++)
    {
        x[i][j]=i+j;
        y[i][j]=2*i+j;
        a[i]=2*i;
    }
	int _SZ_y_2 = 20;
	int _SZ_y_1 = 20;
	int _SZ_x_2 = 20;
	int _SZ_x_1 = 20;
	int *_DEV_y;
	hipMalloc((void**) &_DEV_y, sizeof(int)*_SZ_y_2*_SZ_y_1);
	hipMemcpy(_DEV_y, y, sizeof(int)*_SZ_y_2*_SZ_y_1, hipMemcpyHostToDevice);
	int *_DEV_x;
	hipMalloc((void**) &_DEV_x, sizeof(int)*_SZ_x_2*_SZ_x_1);
	hipMemcpy(_DEV_x, x, sizeof(int)*_SZ_x_2*_SZ_x_1, hipMemcpyHostToDevice);
	float _NUM_THREADS = 400,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=20;
		_THREADS.y=20;
	}
	else {
		_NUM_BLOCKS=_NUM_THREADS/256;
		_BLOCKS.x=_BLOCKS.y=ceil(sqrt(_NUM_BLOCKS));
		_THREADS.x=_THREADS.y=ceil(sqrt(400.0/(_BLOCKS.x*_BLOCKS.y)));
		int temp=_NUM_BLOCKS;
		if(_NUM_BLOCKS>_NBLOCK)
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
	}
	int _CUDA_TILE;
	for(i=0;i<=20;i+=3)
	for(j=0;j<=15;j+=2)
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_y, _SZ_y_2, _SZ_y_1, _DEV_x, _SZ_x_2, _SZ_x_1, 2, i, j, 0, 20, 0, 15, _CUDA_TILE);
	hipDeviceSynchronize();
	hipMemcpy(y, _DEV_y, sizeof(int)*_SZ_y_2*_SZ_y_1, hipMemcpyDeviceToHost);
	hipMemcpy(x, _DEV_x, sizeof(int)*_SZ_x_2*_SZ_x_1, hipMemcpyDeviceToHost);
    for(i=0;i<20;i++)
    for(j=0;j<20;j++)
        printf("%d\t%d\n",x[i][j],y[i][j]);
    return 0;
}



__global__ void _AFFINE_KERNEL(int* y,int _SZ_y_2,int _SZ_y_1,int* x,int _SZ_x_2,int _SZ_x_1,int phi_count, int CUDA_i, int CUDA_j, int CUDA_L_i,int CUDA_U_i, int CUDA_L_j,int CUDA_U_j, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	int j = gridDim.y*blockDim.y*_CUDA_TILE + blockDim.y*blockIdx.y + threadIdx.y;
	if((CUDA_i<=i)&&(i<(CUDA_i+3))&&(i<=CUDA_U_i)){
	if((CUDA_j<=j)&&(j<(CUDA_j+2))&&(j<=CUDA_U_j)){
		x[(3+i+5)*_SZ_x_1+5+j+1]=y[(3+i-1)*_SZ_y_1+5+j-2]+x[(3+i)*_SZ_x_1+5+j-1];
		y[(3+i+2)*_SZ_y_1+5+j+2]=x[(3+i+2)*_SZ_x_1+5+j-1];
}}}

