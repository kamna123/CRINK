__global__ void _AFFINE_KERNEL(int* ,int ,int ,int ,int ,int);

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
int main(int argc, char** argv)
{
    int N = 1000;
	int N_NODES = 100;
	int data = 1;
	int _NTHREAD = 1, _NBLOCK = 1;
	char* readfile, *outfile;
	if(argc>1) _NTHREAD = atoi(argv[1]);
	if(argc>2) _NBLOCK = atoi(argv[2]);
	if(argc>3) data = atoi(argv[3]) + 1;
	if(argc>4) readfile = argv[4];
	int i,j;
	//srand(time(NULL));
	FILE* f;
	f = fopen(readfile, "r");
	j=0;
	char c;
	while(1){
		c = fgetc(f);
		if(c=='\n') { 
			j++;
			c = fgetc(f);
			if(c!='%') break;
		}
	}
	fscanf(f, "%d", &N_NODES);
	//printf("---------%d  ----------",N_NODES);
	fscanf(f, "%d", &N_NODES);
	fscanf(f, "%d", &N);
	if(2*N<_NTHREAD*_NBLOCK) {
		printf("%d\n",_NTHREAD*_NBLOCK);
		fclose(f);
		return 0;
	}
	struct timespec start, end, mid_start, mid_end;
	double runTime, pre_time, post_time, computeTime;
    outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+10));
	strcpy(outfile, readfile);
	strcat(outfile, ".data");
	FILE* fp;
	fp = fopen(outfile, "a");
    int q[N],x;    
    for (i = 0; i < N; i++) 
        {
		    fscanf(f, "%d", &q[i]);
		    fscanf(f, "%d", &x);
	    }
    clock_gettime(CLOCK_MONOTONIC, &start);
	int _SZ_q_1 = N;
	int *_DEV_q;
	hipMalloc((void**) &_DEV_q, sizeof(int)*_SZ_q_1);
	hipMemcpy(_DEV_q, q, sizeof(int)*_SZ_q_1, hipMemcpyHostToDevice);
	int _NUM_THREADS = N,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=_NUM_THREADS;
	}
	else {
		 _THREADS.x=_NTHREAD;
		_NUM_BLOCKS=(_NUM_THREADS % _NTHREAD == 0)?(_NUM_THREADS/_NTHREAD):((_NUM_THREADS/_NTHREAD)+1);
		if(_NUM_BLOCKS<_NBLOCK)
			_BLOCKS.x=_NUM_BLOCKS;
		else {
			_BLOCKS.x=_NBLOCK;
			int temp=_NUM_BLOCKS;
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
		}
	}
	int ID_1, ID_2, START[1];
	int _CUDA_TILE;
	int Phi[1]={2};
	int loopUpperLimits[1]={N};
	clock_gettime(CLOCK_MONOTONIC, &mid_start);
	for(ID_1=1;ID_1<=N/2+1;ID_1++)
	{
		for(ID_2=0;ID_2<1;ID_2++)
		{
			if(Phi[ID_2]>=0)
				START[ID_2]=(ID_1-1)*Phi[ID_2];
			else
				START[ID_2]=loopUpperLimits[ID_2]+(ID_1-1)*Phi[ID_2];
		}
		for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		{
			_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_q, _SZ_q_1, START[0], MIN(START[0]+2, N), _CUDA_TILE, N);
			hipDeviceSynchronize();
		}
	}
	clock_gettime(CLOCK_MONOTONIC, &mid_end);
	hipMemcpy(q, _DEV_q, sizeof(int)*_SZ_q_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_q);
    clock_gettime(CLOCK_MONOTONIC, &end);
	pre_time = (double) ((((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	post_time = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec)) / 1000000000;
	computeTime = (double) ((((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec) - (((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec)) / 1000000000;
	
	runTime = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	printf("********************************\n");
	fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS.x,_BLOCKS.x,data,pre_time,computeTime,post_time,runTime,_CUDA_TILE);
	//fprintf(fp,"%d,%d,%.14f\n",N_EDGES,data,runTime);
	//fclose(fp);
	printf("RUN TIME: %.14f\n", runTime);

	fclose(fp);
	fclose(f);
    return 0;
}



__global__ void _AFFINE_KERNEL(int* q,int _SZ_q_1,int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE, int N)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
		q[N-i]=q[N-i+2];
}}

