#include "hip/hip_runtime.h"
#define _NTHREAD 512
#define _NBLOCK 65535
#include<hip/hip_runtime.h>

__global__ void _AFFINE_KERNEL(int* ,int ,int ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
int main(int argc, char** argv)
{
    int N = 1000;
	int N_NODES = 100;
	int data = 1;
	//int _NTHREAD = 1, _NBLOCK = 1;
	char* readfile, *outfile;
	if(argc>1) _NTHREAD = atoi(argv[1]);
	if(argc>2) _NBLOCK = atoi(argv[2]);
	if(argc>3) data = atoi(argv[3]) + 1;
	if(argc>4) readfile = argv[4];
	int i,j;
	//srand(time(NULL));
	FILE* f;
	f = fopen(readfile, "r");
	j=0;
	char c;
	while(1){
		c = fgetc(f);
		if(c=='\n') { 
			j++;
			c = fgetc(f);
			if(c!='%') break;
		}
	}
	fscanf(f, "%d", &N_NODES);
	//printf("---------%d  ----------",N_NODES);
	fscanf(f, "%d", &N_NODES);
	fscanf(f, "%d", &N);
	if(2*N<_NTHREAD*_NBLOCK) {
		printf("%d\n",_NTHREAD*_NBLOCK);
		fclose(f);
		return 0;
	}
	struct timespec start, end, mid_start, mid_end;
	double runTime, pre_time, post_time, computeTime;
    int a[20],i,j,k;
    int n=20;
    for(i=0;i<20;i++)
    {
        a[i]=2*i;
    }
    clock_gettime(CLOCK_MONOTONIC, &start);
	int _SZ_a_1 = 20;
	int *_DEV_a;
	hipMalloc((void**) &_DEV_a, sizeof(int)*_SZ_a_1);
	hipMemcpy(_DEV_a, a, sizeof(int)*_SZ_a_1, hipMemcpyHostToDevice);
	float _NUM_THREADS = 20,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=_NUM_THREADS;
	}
	else {
		 _THREADS.x=_NTHREAD;
		_NUM_BLOCKS=_NUM_THREADS/512;
		if(_NUM_BLOCKS<_NBLOCK)
			_BLOCKS.x=_NUM_BLOCKS;
		else {
			_BLOCKS.x=_NBLOCK;
			int temp=_NUM_BLOCKS;
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
		}
	}
	int _CUDA_TILE;
	for(i=0;i<18;i+=2)
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_a, _SZ_a_1, 1, i, 0, 18, _CUDA_TILE);
	hipDeviceSynchronize();
	hipMemcpy(a, _DEV_a, sizeof(int)*_SZ_a_1, hipMemcpyDeviceToHost);
    clock_gettime(CLOCK_MONOTONIC, &end);
	pre_time = (double) ((((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	post_time = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec)) / 1000000000;
	computeTime = (double) ((((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec) - (((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec)) / 1000000000;
	
	runTime = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	printf("********************************\n");
	fprintf(fp,"%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,data,pre_time,computeTime,post_time,runTime,_CUDA_TILE);
	//fprintf(fp,"%d,%d,%.14f\n",N_EDGES,data,runTime);
	//fclose(fp);
	printf("RUN TIME: %.14f\n", runTime);

	fclose(fp);
	fclose(f);
    return 0;
}



__global__ void _AFFINE_KERNEL(int* a,int _SZ_a_1,int phi_count, int CUDA_i, int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_i<=i)&&(i<(CUDA_i+2))&&(i<CUDA_U_i)){
		a[18-i+1]=a[18-i-1];
}}

