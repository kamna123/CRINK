#define _NTHREAD 512
#define _NBLOCK 65535
#include<hip/hip_runtime.h>

__global__ void _AFFINE_KERNEL(int* ,int ,int* ,int ,int ,int ,int ,int ,int ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
int main()
{
    int XP1[20][20],XS3[20],i,j,k;
    for(i=0;i<20;i++)
    for(j=0;j<20;j++)
    {
        XP1[i][j]=i+j;
        XS3[i]=2*i;
    }
	int _SZ_XS3_1 = 20;
	int _SZ_XP1_2 = 20;
	int _SZ_XP1_1 = 20;
	int *_DEV_XS3;
	hipMalloc((void**) &_DEV_XS3, sizeof(int)*_SZ_XS3_1);
	hipMemcpy(_DEV_XS3, XS3, sizeof(int)*_SZ_XS3_1, hipMemcpyHostToDevice);
	int *_DEV_XP1;
	hipMalloc((void**) &_DEV_XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1);
	hipMemcpy(_DEV_XP1, XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1, hipMemcpyHostToDevice);
	float _NUM_THREADS = 400,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=20;
		_THREADS.y=20;
	}
	else {
		_NUM_BLOCKS=_NUM_THREADS/256;
		_BLOCKS.x=_BLOCKS.y=ceil(sqrt(_NUM_BLOCKS));
		_THREADS.x=_THREADS.y=ceil(sqrt(400.0/(_BLOCKS.x*_BLOCKS.y)));
		int temp=_NUM_BLOCKS;
		if(_NUM_BLOCKS>_NBLOCK)
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
	}
	int _CUDA_TILE;
	for(i=0;i<20;i+=3)
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_XS3, _SZ_XS3_1, _DEV_XP1, _SZ_XP1_2, _SZ_XP1_1, 2, i, j, 0, 20, 0, 20, _CUDA_TILE);
	hipDeviceSynchronize();
	hipMemcpy(XP1, _DEV_XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1, hipMemcpyDeviceToHost);
    return 0;
}



__global__ void _AFFINE_KERNEL(int* XS3,int _SZ_XS3_1,int* XP1,int _SZ_XP1_2,int _SZ_XP1_1,int phi_count, int CUDA_i, int CUDA_j, int CUDA_L_i,int CUDA_U_i, int CUDA_L_j,int CUDA_U_j, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	int j = gridDim.y*blockDim.y*_CUDA_TILE + blockDim.y*blockIdx.y + threadIdx.y;
	if((CUDA_i<=i)&&(i<(CUDA_i+3))&&(i<CUDA_U_i)){
	if((CUDA_L_j<=j)&&(j<CUDA_U_j)){
		XP1[i*_SZ_XP1_1+j]=XP1[(i+3)*_SZ_XP1_1+j+4]+XS3[i];
}}}

