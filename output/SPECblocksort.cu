#define _NTHREAD 512
#define _NBLOCK 65535
#include<hip/hip_runtime.h>

__global__ void _AFFINE_KERNEL(int* ,int ,int* ,int ,int ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
int main()
{
    int block[20],quadrant[20],i,j,k;
    for(i=0;i<20;i++)
    {
        block[i]=2*i;
        quadrant[i]=3*i;
    }
	int _SZ_quadrant_1 = 20;
	int _SZ_block_1 = 20;
	int *_DEV_quadrant;
	hipMalloc((void**) &_DEV_quadrant, sizeof(int)*_SZ_quadrant_1);
	hipMemcpy(_DEV_quadrant, quadrant, sizeof(int)*_SZ_quadrant_1, hipMemcpyHostToDevice);
	int *_DEV_block;
	hipMalloc((void**) &_DEV_block, sizeof(int)*_SZ_block_1);
	hipMemcpy(_DEV_block, block, sizeof(int)*_SZ_block_1, hipMemcpyHostToDevice);
	int _NUM_THREADS = 20;
	float _NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=_NUM_THREADS;
	}
	else {
		 _THREADS.x=_NTHREAD;
		_NUM_BLOCKS=(_NUM_THREADS % _NTHREAD == 0)?(_NUM_THREADS/_NTHREAD):((_NUM_THREADS/_NTHREAD)+1);
		if(_NUM_BLOCKS<_NBLOCK)
			_BLOCKS.x=_NUM_BLOCKS;
		else {
			_BLOCKS.x=_NBLOCK;
			int temp=_NUM_BLOCKS;
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
		}
	}
	int _CUDA_TILE;
	for(i=0;i<20;i+=15)
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
	{		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_quadrant, _SZ_quadrant_1, _DEV_block, _SZ_block_1, 1, i, 0, 20, _CUDA_TILE);
		hipDeviceSynchronize();
	}	hipMemcpy(block, _DEV_block, sizeof(int)*_SZ_block_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_block);
}



__global__ void _AFFINE_KERNEL(int* quadrant,int _SZ_quadrant_1,int* block,int _SZ_block_1,int phi_count, int CUDA_i, int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_i<=i)&&(i<(CUDA_i+15))&&(i<CUDA_U_i)){
		block[15+i]=block[i];
		quadrant[15+i]=0;
}}

