#include<hip/hip_runtime.h>
#include<time.h>
__global__ void _AFFINE_KERNEL(int* ,int ,int* ,int ,int ,int ,int ,int ,int ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
int main(int argc, char** argv)
{
    int N = 1000;
	int N_NODES = 100;
	int data = 1;
	int _NTHREAD = 1, _NBLOCK = 1;
	char* readfile, *outfile;
	if(argc>1) _NTHREAD = atoi(argv[1]);
	if(argc>2) _NBLOCK = atoi(argv[2]);
	if(argc>3) data = atoi(argv[3]) + 1;
	if(argc>4) readfile = argv[4];
	int i,j;
	FILE* f;
	f = fopen(readfile, "r");
	j=0;
	char c;
	while(1){
		c = fgetc(f);
		if(c=='\n') { 
			j++;
			c = fgetc(f);
			if(c!='%') break;
		}
	}
	fscanf(f, "%d", &N_NODES);
	fscanf(f, "%d", &N_NODES);
	fscanf(f, "%d", &N);
	if(2*N<_NTHREAD*_NBLOCK) {
		printf("%d\n",_NTHREAD*_NBLOCK);
		fclose(f);
		return 0;
	}
	struct timespec start, end, mid_start, mid_end;
	double runTime, pre_time, post_time, computeTime;
    outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
	strcpy(outfile, readfile);
	strcat(outfile, ".data_Higbie1_TD");
	FILE* fp;
	fp = fopen(outfile, "a");
    int M,I; 
    M=sqrt(2.0*N); 
    int XP1[M][M],XS3[N];  
    i=0;
    j=0;
    for (I = 0; I < N; I++) 
    {
        if((j<=M-1)&&(i<=M-1))
        {
            fscanf(f, "%d", &XP1[i][j++]);
            XS3[I]=XP1[i][j-1];
            if(j<M)    fscanf(f, "%d", &XP1[i][j++]);
            else if(i<M-1)
            {
                j=0;
                fscanf(f, "%d", &XP1[++i][j++]);
            }
        }
        else if((j==M)&&(i<M-1))
        {
            j=0;
            fscanf(f, "%d", &XP1[++i][j++]);
            XS3[I]=XP1[i][j-1];
            fscanf(f, "%d", &XP1[i][j++]);
        }
        else if(i==M-1)
		    fscanf(f, "%d", &XS3[I]);
	}
    clock_gettime(CLOCK_MONOTONIC, &start);
	int _SZ_XS3_1 = N;
	int _SZ_XP1_2 = M;
	int _SZ_XP1_1 = M;
	int *_DEV_XS3;
	hipMalloc((void**) &_DEV_XS3, sizeof(int)*_SZ_XS3_1);
	hipMemcpy(_DEV_XS3, XS3, sizeof(int)*_SZ_XS3_1, hipMemcpyHostToDevice);
	int *_DEV_XP1;
	hipMalloc((void**) &_DEV_XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1);
	hipMemcpy(_DEV_XP1, XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1, hipMemcpyHostToDevice);
	float _NUM_THREADS = M*M,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=M;
		_THREADS.y=M;
	}
	else {
		_NUM_BLOCKS=_NUM_THREADS/256;
		_BLOCKS.x=_BLOCKS.y=ceil(sqrt(_NUM_BLOCKS));
		_THREADS.x=_THREADS.y=ceil(sqrt((M*M*1.0)/(_BLOCKS.x*_BLOCKS.y)));
		int temp=_NUM_BLOCKS;
		if(_NUM_BLOCKS>_NBLOCK)
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
	}
	int _CUDA_TILE;
	clock_gettime(CLOCK_MONOTONIC, &mid_start);
	for(i=0;i<20;i+=3)
	for(j=0;j<20;j+=4)
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
	{
		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_XS3, _SZ_XS3_1, _DEV_XP1, _SZ_XP1_2, _SZ_XP1_1, 2, i, j, 0, 20, 0, 20, _CUDA_TILE);
	    hipDeviceSynchronize();
	}
	clock_gettime(CLOCK_MONOTONIC, &mid_end);
	hipMemcpy(XP1, _DEV_XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_XP1);
    clock_gettime(CLOCK_MONOTONIC, &end);
	pre_time = (double) ((((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	post_time = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec)) / 1000000000;
	computeTime = (double) ((((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec) - (((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec)) / 1000000000;
	
	runTime = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	printf("********************************\n");
	fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS.x,_BLOCKS.x,data,pre_time,computeTime,post_time,runTime,_CUDA_TILE);
	printf("RUN TIME: %.14f\n", runTime);

	fclose(fp);
	fclose(f);
    return 0;
}



__global__ void _AFFINE_KERNEL(int* XS3,int _SZ_XS3_1,int* XP1,int _SZ_XP1_2,int _SZ_XP1_1,int phi_count, int CUDA_i, int CUDA_j, int CUDA_L_i,int CUDA_U_i, int CUDA_L_j,int CUDA_U_j, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	int j = gridDim.y*blockDim.y*_CUDA_TILE + blockDim.y*blockIdx.y + threadIdx.y;
	if((CUDA_i<=i)&&(i<(CUDA_i+3))&&(i<CUDA_U_i)){
	if((CUDA_j<=j)&&(j<(CUDA_j+4))&&(j<CUDA_U_j)){
		XP1[i*_SZ_XP1_1+j]=XP1[(i+3)*_SZ_XP1_1+j+4]+XS3[i];
}}}

