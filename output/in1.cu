#include "hip/hip_runtime.h"
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_0(int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_1(int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
int main()
{
    int x[20],y[20],i,j,k;
     // PARALLELIZE IN CUDA START
    for(i=0;i<20;i++)
    {
        x[i]=28*i;
    }
     // PARALLELIZE IN CUDA END
   // PARALLELIZE IN CUDA START
    for(i=5;i<=19;i++)
    {
       x[i]=y[i-5];
       y[i]=x[i-4];
       //y[2*i-3][3*j-5]=x[i][j];
       //x[3*i-2][2*j-4]=y[i][j];
     } 
    // PARALLELIZE IN CUDA END
    return 0;
}


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_y_0 = 20;
	int _SZ_x_0 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_y_0;
	hipMalloc((void**) &_DEV_y_0, sizeof(int)*_SZ_y_0);
	hipMemcpy(_DEV_y_0, y, sizeof(int)*_SZ_y_0, hipMemcpyHostToDevice);
	int *_DEV_x_0;
	hipMalloc((void**) &_DEV_x_0, sizeof(int)*_SZ_x_0);
	hipMemcpy(_DEV_x_0, x, sizeof(int)*_SZ_x_0, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_0 = 20;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int ID_1_0, ID_2_0, START_0[1];
	int _CUDA_TILE_0;
	int Phi_0[1]={4};
	int loopUpperLimits_0[1]={20};
	for(ID_1_0=1;ID_1_0<=20/4+1;ID_1_0++)
	{
		for(ID_2_0=0;ID_2_0<1;ID_2_0++)
		{
			if(Phi_0[ID_2_0]>=0)
				START_0[ID_2_0]=(ID_1_0-1)*Phi_0[ID_2_0];
			else
				START_0[ID_2_0]=loopUpperLimits_0[ID_2_0]+(ID_1_0-1)*Phi_0[ID_2_0];
		}
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>(_DEV_y_0, _SZ_y_0, _DEV_x_0, _SZ_x_0, START_0[0], MIN(START_0[0]+4, 20), _CUDA_TILE_0);
			hipDeviceSynchronize();
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif

	// ---------Copying Kernel variable from device to host----------

	// ---------Releasing the memory allocated to kernel variable----------

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);
		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_0);
	#endif
	#endif

	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_1, end_1, mid_start_1, mid_end_1;
		double runTime_1, pre_time_1, post_time_1, computeTime_1;
	#endif
	int _SZ_y_1 = 20;
	int _SZ_x_1 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_1);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_y_1;
	hipMalloc((void**) &_DEV_y_1, sizeof(int)*_SZ_y_1);
	hipMemcpy(_DEV_y_1, y, sizeof(int)*_SZ_y_1, hipMemcpyHostToDevice);
	int *_DEV_x_1;
	hipMalloc((void**) &_DEV_x_1, sizeof(int)*_SZ_x_1);
	hipMemcpy(_DEV_x_1, x, sizeof(int)*_SZ_x_1, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_1 = 20;
	float _NUM_BLOCKS_1=1;
	int _NUM_TILE_1=1;
	dim3 _THREADS_1(512);
	dim3 _BLOCKS_1(1);
	if(_NUM_THREADS_1 < _NTHREAD)
	{
		_THREADS_1.x=_NUM_THREADS_1;
	}
	else {
		 _THREADS_1.x=_NTHREAD;
		_NUM_BLOCKS_1=(_NUM_THREADS_1 % _NTHREAD == 0)?(_NUM_THREADS_1/_NTHREAD):((_NUM_THREADS_1/_NTHREAD)+1);
		if(_NUM_BLOCKS_1<_NBLOCK)
			_BLOCKS_1.x=_NUM_BLOCKS_1;
		else {
			_BLOCKS_1.x=_NBLOCK;
			int temp_1=_NUM_BLOCKS_1;
			_NUM_TILE_1=(temp_1 % _NBLOCK == 0)?(_NUM_BLOCKS_1/_NBLOCK):((_NUM_BLOCKS_1/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_1);
	#endif

	int ID_1_1, ID_2_1, START_1[1];
	int _CUDA_TILE_1;
	int Phi_1[1]={4};
	int loopUpperLimits_1[1]={14};
	for(ID_1_1=1;ID_1_1<=14/4+1;ID_1_1++)
	{
		for(ID_2_1=0;ID_2_1<1;ID_2_1++)
		{
			if(Phi_1[ID_2_1]>=0)
				START_1[ID_2_1]=(ID_1_1-1)*Phi_1[ID_2_1];
			else
				START_1[ID_2_1]=loopUpperLimits_1[ID_2_1]+(ID_1_1-1)*Phi_1[ID_2_1];
		}
	for(_CUDA_TILE_1=0;_CUDA_TILE_1<_NUM_TILE_1;_CUDA_TILE_1++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_1<<<_BLOCKS_1,_THREADS_1>>>(_DEV_y_1, _SZ_y_1, _DEV_x_1, _SZ_x_1, START_1[0], MIN(START_1[0]+4, 14), _CUDA_TILE_1);
			hipDeviceSynchronize();
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_1);
	#endif

	// ---------Copying Kernel variable from device to host----------
	hipMemcpy(y, _DEV_y_1, sizeof(int)*_SZ_y_1, hipMemcpyDeviceToHost);
	hipMemcpy(x, _DEV_x_1, sizeof(int)*_SZ_x_1, hipMemcpyDeviceToHost);

	// ---------Releasing the memory allocated to kernel variable----------
	hipFree(_DEV_y_1);
	hipFree(_DEV_x_1);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_1);
		pre_time_1 = (double) ((((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
		post_time_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec)) / 1000000000;
		computeTime_1 = (double) ((((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec) - (((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec)) / 1000000000;
		runTime_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_1.x,_BLOCKS_1.x,data,pre_time_1,computeTime_1,post_time_1,runTime_1,_CUDA_TILE_1);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_1);
	#endif
	#endif
}

// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_0(int* y,int _SZ_y_0,int* x,int _SZ_x_0,int CUDA_L_i_0,int CUDA_U_i_0, int _CUDA_TILE_0)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_0 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i_0<=i)&&(i<=CUDA_U_i_0)){
		x[i]=28*i;
}}



// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_1(int* y,int _SZ_y_1,int* x,int _SZ_x_1,int CUDA_L_i_1,int CUDA_U_i_1, int _CUDA_TILE_1)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_1 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i_1<=i)&&(i<=CUDA_U_i_1)){
		x[5+i]=y[5+i-5];
		y[5+i]=x[5+i-4];
}}

