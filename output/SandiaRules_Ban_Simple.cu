#include "hip/hip_runtime.h"
// NO DEPENDENCY case of SandiaRules.c
__global__ void _AFFINE_KERNEL(int* ,int ,int* ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
#include<time.h>
int main(int argc, char** argv){
{
    int N = 1000;
	int N_NODES = 100;
	int data = 1;
	int _NTHREAD = 1, _NBLOCK = 1;
	char* readfile, *outfile;
	if(argc>1) _NTHREAD = atoi(argv[1]);
	if(argc>2) _NBLOCK = atoi(argv[2]);
	if(argc>3) data = atoi(argv[3]) + 1;
	if(argc>4) readfile = argv[4];
	int i,j;
	FILE* f;
	f = fopen(readfile, "r");
	j=0;
	char c;
	while(1){
		c = fgetc(f);
		if(c=='\n') { 
			j++;
			c = fgetc(f);
			if(c!='%') break;
		}
	}
	fscanf(f, "%d", &N_NODES);
	fscanf(f, "%d", &N_NODES);
	fscanf(f, "%d", &N);
	if(2*N<_NTHREAD*_NBLOCK) {
		printf("%d\n",_NTHREAD*_NBLOCK);
		fclose(f);
		return 0;
	}
	struct timespec start, end, mid_start, mid_end;
	double runTime, pre_time, post_time, computeTime;
    outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
	strcpy(outfile, readfile);
	strcat(outfile, ".data_Sandia_Ban_Simple");
	FILE* fp;
	fp = fopen(outfile, "a");
    int x[N],w[N];    
    for (i = 0; i < N; i++) 
        {
		    fscanf(f, "%d", &x[i]);
		    fscanf(f, "%d", &w[i]);
	    }
    clock_gettime(CLOCK_MONOTONIC, &start);
	int _SZ_w_1 = N;
	int _SZ_x_1 = N;
	int *_DEV_w;
	hipMalloc((void**) &_DEV_w, sizeof(int)*_SZ_w_1);
	hipMemcpy(_DEV_w, w, sizeof(int)*_SZ_w_1, hipMemcpyHostToDevice);
	int *_DEV_x;
	hipMalloc((void**) &_DEV_x, sizeof(int)*_SZ_x_1);
	hipMemcpy(_DEV_x, x, sizeof(int)*_SZ_x_1, hipMemcpyHostToDevice);
	int _NUM_THREADS = N,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=_NUM_THREADS;
	}
	else {
		 _THREADS.x=_NTHREAD;
		_NUM_BLOCKS=_NUM_THREADS/512;
		if(_NUM_BLOCKS<_NBLOCK)
			_BLOCKS.x=_NUM_BLOCKS;
		else {
			_BLOCKS.x=_NBLOCK;
			int temp=_NUM_BLOCKS;
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
		}
	}
	int _CUDA_TILE;
	clock_gettime(CLOCK_MONOTONIC, &mid_start);
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
	{
		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_w, _SZ_w_1, _DEV_x, _SZ_x_1, 0, N, _CUDA_TILE);
	    hipDeviceSynchronize();
	    }
	clock_gettime(CLOCK_MONOTONIC, &mid_end);
	hipMemcpy(w, _DEV_w, sizeof(int)*_SZ_w_1, hipMemcpyDeviceToHost);
	hipMemcpy(x, _DEV_x, sizeof(int)*_SZ_x_1, hipMemcpyDeviceToHost);
    hipFree(_DEV_w);
    hipFree(_DEV_x);
    clock_gettime(CLOCK_MONOTONIC, &end);
	pre_time = (double) ((((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	post_time = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec)) / 1000000000;
	computeTime = (double) ((((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec) - (((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec)) / 1000000000;
	
	runTime = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	printf("********************************\n");
	fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS.x,_BLOCKS.x,data,pre_time,computeTime,post_time,runTime,_CUDA_TILE);
	printf("RUN TIME: %.14f\n", runTime);

	fclose(fp);
	fclose(f);
    return 0;
}



__global__ void _AFFINE_KERNEL(int* w,int _SZ_w_1,int* x,int _SZ_x_1,int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
		x[1+i-1]=-x[CUDA_U_i-1-i];
		w[1+i-1]=w[CUDA_U_i-1-i];
}}

