#include "hip/hip_runtime.h"
#define _NTHREAD 512
#define _NBLOCK 65535

__global__ void _AFFINE_KERNEL(int ,int ,int ,int ,int ,int ,int ,int ,int* ,int ,int ,int ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
int main()
{
    int px[20][20],i,j,l,dm28,dm25,dm22,dm27,dm23,dm24,dm26,c0;
    for(i=0;i<20;i++)
    for(j=0;j<20;j++)
    {
        px[i][j]=i+j;
    }
    dm22=1,dm23=2,dm24=3,dm24=4,dm25=5,dm26=6,dm
	int _SZ_px_2 = 20;
	int _SZ_px_1 = 20;
	int *_DEV_px;
	hipMalloc((void**) &_DEV_px, sizeof(int)*_SZ_px_2*_SZ_px_1);
	hipMemcpy(_DEV_px, px, sizeof(int)*_SZ_px_2*_SZ_px_1, hipMemcpyHostToDevice);
	int _NUM_THREADS = 400;
	float _NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=20;
		_THREADS.y=20;
	}
	else {
		_NUM_BLOCKS=(_NUM_THREADS*1.0)/256;
		_BLOCKS.x=_BLOCKS.y=ceil(sqrt(_NUM_BLOCKS));
		_THREADS.x=_THREADS.y=ceil(sqrt(400.0/(_BLOCKS.x*_BLOCKS.y)));
		int temp=_NUM_BLOCKS;
		if(_NUM_BLOCKS>_NBLOCK)
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
	}
	int _CUDA_TILE;
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
	{		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(c0, dm22, dm23, dm24, dm25, dm26, dm27, dm28, _DEV_px, _SZ_px_2, _SZ_px_1, 0, 20, 0, 20, _CUDA_TILE);
		hipDeviceSynchronize();
	}
	hipMemcpy(px, _DEV_px, sizeof(int)*_SZ_px_2*_SZ_px_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_px);
    return 0;
}


__global__ void _AFFINE_KERNEL(int c0,int dm22,int dm23,int dm24,int dm25,int dm26,int dm27,int dm28,int* px,int _SZ_px_2,int _SZ_px_1,int CUDA_L_l,int CUDA_U_l, int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE)
{
	int l = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	int i = gridDim.y*blockDim.y*_CUDA_TILE + blockDim.y*blockIdx.y + threadIdx.y;
	if((CUDA_L_l<=l)&&(l<=CUDA_U_l)){
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
		px[i*_SZ_px_1+0]=dm28*px[i*_SZ_px_1+12]+dm27*px[i*_SZ_px_1+11]+dm26*px[i*_SZ_px_1+10]+dm25*px[i*_SZ_px_1+9]+dm24*px[i*_SZ_px_1+8]+dm23*px[i*_SZ_px_1+7]+dm22*px[i*_SZ_px_1+6]+c0*(px[i*_SZ_px_1+4]+px[i*_SZ_px_1+5])+px[i*_SZ_px_1+2];
}}}

