#define _NTHREAD 512
#define _NBLOCK 65535

__global__ void _AFFINE_KERNEL(int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
int main()
{
    int x[20];
    int w[20],i,j,k;
    for(i=0;i<20;i++)
    {
        x[i]=2*i;
        w[i]=2*i;
    }
	int _SZ_w_1 = 20;
	int _SZ_x_1 = 20;
	int *_DEV_w;
	hipMalloc((void**) &_DEV_w, sizeof(int)*_SZ_w_1);
	hipMemcpy(_DEV_w, w, sizeof(int)*_SZ_w_1, hipMemcpyHostToDevice);
	int *_DEV_x;
	hipMalloc((void**) &_DEV_x, sizeof(int)*_SZ_x_1);
	hipMemcpy(_DEV_x, x, sizeof(int)*_SZ_x_1, hipMemcpyHostToDevice);
	int _NUM_THREADS = 20;
	float _NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=_NUM_THREADS;
	}
	else {
		 _THREADS.x=_NTHREAD;
		_NUM_BLOCKS=(_NUM_THREADS % _NTHREAD == 0)?(_NUM_THREADS/_NTHREAD):((_NUM_THREADS/_NTHREAD)+1);
		if(_NUM_BLOCKS<_NBLOCK)
			_BLOCKS.x=_NUM_BLOCKS;
		else {
			_BLOCKS.x=_NBLOCK;
			int temp=_NUM_BLOCKS;
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
		}
	}
	int ID_1, ID_2, START[1];
	int _CUDA_TILE;
	int Phi[1]={19};
	int loopUpperLimits[1]={9};
	for(ID_1=1;ID_1<=9/19+1;ID_1++)
	{
		for(ID_2=0;ID_2<1;ID_2++)
		{
			if(Phi[ID_2]>=0)
				START[ID_2]=(ID_1-1)*Phi[ID_2];
			else
				START[ID_2]=loopUpperLimits[ID_2]+(ID_1-1)*Phi[ID_2];
		}
		for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		{
			_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_w, _SZ_w_1, _DEV_x, _SZ_x_1, START[0], MIN(START[0]+19, 9), _CUDA_TILE);
			hipDeviceSynchronize();
		}
	}
	hipMemcpy(w, _DEV_w, sizeof(int)*_SZ_w_1, hipMemcpyDeviceToHost);
	hipMemcpy(x, _DEV_x, sizeof(int)*_SZ_x_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_w);
	hipFree(_DEV_x);
    
    return 0;
}



__global__ void _AFFINE_KERNEL(int* w,int _SZ_w_1,int* x,int _SZ_x_1,int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
		x[1+i-1]=-x[20-1-i];
		w[1+i-1]=w[20-1-i];
}}

