#include "hip/hip_runtime.h"
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_1(int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_2(int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_3(int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
int main()
{
    int a[50],i,x[20],y[20],id[20],j,w[10];
int A[5][5][5];
int B[50][50];
int C[50][50];
int D[50][50];;
    
    for(i=0;i<50;i++)
    {
        a[i]=2*i;
    }
    
     for(i=5;i<=19;i++)
     {
        id[i]=i;
     }
  
    // PARALLELIZE IN CUDA START
for(i=0;i<50;i++){
	
		//A[B[i][j]][C[i][j]][D[i][j]] = (A[C[i][j]][D[i][j]][B[i][j]] + A[D[i][j]][B[i][j]][C[i][j]]) % 1000000007;
              A[a[i]]=A[a[i]]+10;
		
	}


// PARALLELIZE IN CUDA END
  // PARALLELIZE IN CUDA START
    for(i=0;i<46;i++)
    {
       a[i+10]=a[i+2];
     } 
    // PARALLELIZE IN CUDA END
    // PARALLELIZE IN CUDA START
     for ( i = 1; i <=10; i=i+1 )
  {
    w[i] =w[i+1] +3.14 /10;
  }
  // PARALLELIZE IN CUDA END
 // PARALLELIZE IN CUDA START
  for ( i = 1; i <=10; i=i+1 )
  {
    x[i] =x[i+1]+10;
  }
  // PARALLELIZE IN CUDA END
    return 0;
}


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_1, end_1, mid_start_1, mid_end_1;
		double runTime_1, pre_time_1, post_time_1, computeTime_1;
	#endif
	int _SZ_a_1 = 50;
	int _SZ_a_1 = 50;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_1);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_a_1;
	hipMalloc((void**) &_DEV_a_1, sizeof(int)*_SZ_a_1);
	hipMemcpy(_DEV_a_1, a, sizeof(int)*_SZ_a_1, hipMemcpyHostToDevice);
	int *_DEV_a_1;
	hipMalloc((void**) &_DEV_a_1, sizeof(int)*_SZ_a_1);
	hipMemcpy(_DEV_a_1, a, sizeof(int)*_SZ_a_1, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_1 = 50;
	float _NUM_BLOCKS_1=1;
	int _NUM_TILE_1=1;
	dim3 _THREADS_1(512);
	dim3 _BLOCKS_1(1);
	if(_NUM_THREADS_1 < _NTHREAD)
	{
		_THREADS_1.x=_NUM_THREADS_1;
	}
	else {
		 _THREADS_1.x=_NTHREAD;
		_NUM_BLOCKS_1=(_NUM_THREADS_1 % _NTHREAD == 0)?(_NUM_THREADS_1/_NTHREAD):((_NUM_THREADS_1/_NTHREAD)+1);
		if(_NUM_BLOCKS_1<_NBLOCK)
			_BLOCKS_1.x=_NUM_BLOCKS_1;
		else {
			_BLOCKS_1.x=_NBLOCK;
			int temp_1=_NUM_BLOCKS_1;
			_NUM_TILE_1=(temp_1 % _NBLOCK == 0)?(_NUM_BLOCKS_1/_NBLOCK):((_NUM_BLOCKS_1/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_1);
	#endif

	int ID_1_1, ID_2_1, START_1[1];
	int _CUDA_TILE_1;
	int Phi_1[1]={1};
	int loopUpperLimits_1[1]={46};
	for(ID_1_1=1;ID_1_1<=46/1+1;ID_1_1++)
	{
		for(ID_2_1=0;ID_2_1<1;ID_2_1++)
		{
			if(Phi_1[ID_2_1]>=0)
				START_1[ID_2_1]=(ID_1_1-1)*Phi_1[ID_2_1];
			else
				START_1[ID_2_1]=loopUpperLimits_1[ID_2_1]+(ID_1_1-1)*Phi_1[ID_2_1];
		}
	for(_CUDA_TILE_1=0;_CUDA_TILE_1<_NUM_TILE_1;_CUDA_TILE_1++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_1<<<_BLOCKS_1,_THREADS_1>>>(_DEV_a_1, _SZ_a_1, _DEV_a_1, _SZ_a_1, START_1[0], MIN(START_1[0]+1, 46), _CUDA_TILE_1);
			hipDeviceSynchronize();
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_1);
	#endif

	// ---------Copying Kernel variable from device to host----------
	hipMemcpy(a, _DEV_a_1, sizeof(int)*_SZ_a_1, hipMemcpyDeviceToHost);

	// ---------Releasing the memory allocated to kernel variable----------
	hipFree(_DEV_a_1);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_1);
		pre_time_1 = (double) ((((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
		post_time_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec)) / 1000000000;
		computeTime_1 = (double) ((((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec) - (((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec)) / 1000000000;
		runTime_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_1.x,_BLOCKS_1.x,data,pre_time_1,computeTime_1,post_time_1,runTime_1,_CUDA_TILE_1);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_1);
	#endif
	#endif

	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_2, end_2, mid_start_2, mid_end_2;
		double runTime_2, pre_time_2, post_time_2, computeTime_2;
	#endif
	int _SZ_a_2 = 50;
	int _SZ_w_2 = 10;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_2);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_a_2;
	hipMalloc((void**) &_DEV_a_2, sizeof(int)*_SZ_a_2);
	hipMemcpy(_DEV_a_2, a, sizeof(int)*_SZ_a_2, hipMemcpyHostToDevice);
	int *_DEV_w_2;
	hipMalloc((void**) &_DEV_w_2, sizeof(int)*_SZ_w_2);
	hipMemcpy(_DEV_w_2, w, sizeof(int)*_SZ_w_2, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_2 = 50;
	float _NUM_BLOCKS_2=1;
	int _NUM_TILE_2=1;
	dim3 _THREADS_2(512);
	dim3 _BLOCKS_2(1);
	if(_NUM_THREADS_2 < _NTHREAD)
	{
		_THREADS_2.x=_NUM_THREADS_2;
	}
	else {
		 _THREADS_2.x=_NTHREAD;
		_NUM_BLOCKS_2=(_NUM_THREADS_2 % _NTHREAD == 0)?(_NUM_THREADS_2/_NTHREAD):((_NUM_THREADS_2/_NTHREAD)+1);
		if(_NUM_BLOCKS_2<_NBLOCK)
			_BLOCKS_2.x=_NUM_BLOCKS_2;
		else {
			_BLOCKS_2.x=_NBLOCK;
			int temp_2=_NUM_BLOCKS_2;
			_NUM_TILE_2=(temp_2 % _NBLOCK == 0)?(_NUM_BLOCKS_2/_NBLOCK):((_NUM_BLOCKS_2/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_2);
	#endif

	int ID_1_2, ID_2_2, START_2[1];
	int _CUDA_TILE_2;
	int Phi_2[1]={1};
	int loopUpperLimits_2[1]={9};
	for(ID_1_2=1;ID_1_2<=9/1+1;ID_1_2++)
	{
		for(ID_2_2=0;ID_2_2<1;ID_2_2++)
		{
			if(Phi_2[ID_2_2]>=0)
				START_2[ID_2_2]=(ID_1_2-1)*Phi_2[ID_2_2];
			else
				START_2[ID_2_2]=loopUpperLimits_2[ID_2_2]+(ID_1_2-1)*Phi_2[ID_2_2];
		}
	for(_CUDA_TILE_2=0;_CUDA_TILE_2<_NUM_TILE_2;_CUDA_TILE_2++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_2<<<_BLOCKS_2,_THREADS_2>>>(_DEV_a_2, _SZ_a_2, _DEV_w_2, _SZ_w_2, START_2[0], MIN(START_2[0]+1, 9), _CUDA_TILE_2);
			hipDeviceSynchronize();
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_2);
	#endif

	// ---------Copying Kernel variable from device to host----------
	hipMemcpy(w, _DEV_w_2, sizeof(int)*_SZ_w_2, hipMemcpyDeviceToHost);

	// ---------Releasing the memory allocated to kernel variable----------
	hipFree(_DEV_w_2);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_2);
		pre_time_2 = (double) ((((&mid_start_2)->tv_sec * 1000000000) + (&mid_start_2)->tv_nsec) - (((&start_2)->tv_sec * 1000000000) + (&start_2)->tv_nsec)) / 1000000000;
		post_time_2 = (double) ((((&end_2)->tv_sec * 1000000000) + (&end_2)->tv_nsec) - (((&mid_end_2)->tv_sec * 1000000000) + (&mid_end_2)->tv_nsec)) / 1000000000;
		computeTime_2 = (double) ((((&mid_end_2)->tv_sec * 1000000000) + (&mid_end_2)->tv_nsec) - (((&mid_start_2)->tv_sec * 1000000000) + (&mid_start_2)->tv_nsec)) / 1000000000;
		runTime_2 = (double) ((((&end_2)->tv_sec * 1000000000) + (&end_2)->tv_nsec) - (((&start_2)->tv_sec * 1000000000) + (&start_2)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_2.x,_BLOCKS_2.x,data,pre_time_2,computeTime_2,post_time_2,runTime_2,_CUDA_TILE_2);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_2);
	#endif
	#endif

	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_3, end_3, mid_start_3, mid_end_3;
		double runTime_3, pre_time_3, post_time_3, computeTime_3;
	#endif
	int _SZ_a_3 = 50;
	int _SZ_x_3 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_3);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_a_3;
	hipMalloc((void**) &_DEV_a_3, sizeof(int)*_SZ_a_3);
	hipMemcpy(_DEV_a_3, a, sizeof(int)*_SZ_a_3, hipMemcpyHostToDevice);
	int *_DEV_x_3;
	hipMalloc((void**) &_DEV_x_3, sizeof(int)*_SZ_x_3);
	hipMemcpy(_DEV_x_3, x, sizeof(int)*_SZ_x_3, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_3 = 50;
	float _NUM_BLOCKS_3=1;
	int _NUM_TILE_3=1;
	dim3 _THREADS_3(512);
	dim3 _BLOCKS_3(1);
	if(_NUM_THREADS_3 < _NTHREAD)
	{
		_THREADS_3.x=_NUM_THREADS_3;
	}
	else {
		 _THREADS_3.x=_NTHREAD;
		_NUM_BLOCKS_3=(_NUM_THREADS_3 % _NTHREAD == 0)?(_NUM_THREADS_3/_NTHREAD):((_NUM_THREADS_3/_NTHREAD)+1);
		if(_NUM_BLOCKS_3<_NBLOCK)
			_BLOCKS_3.x=_NUM_BLOCKS_3;
		else {
			_BLOCKS_3.x=_NBLOCK;
			int temp_3=_NUM_BLOCKS_3;
			_NUM_TILE_3=(temp_3 % _NBLOCK == 0)?(_NUM_BLOCKS_3/_NBLOCK):((_NUM_BLOCKS_3/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_3);
	#endif

	int ID_1_3, ID_2_3, START_3[1];
	int _CUDA_TILE_3;
	int Phi_3[1]={1};
	int loopUpperLimits_3[1]={9};
	for(ID_1_3=1;ID_1_3<=9/1+1;ID_1_3++)
	{
		for(ID_2_3=0;ID_2_3<1;ID_2_3++)
		{
			if(Phi_3[ID_2_3]>=0)
				START_3[ID_2_3]=(ID_1_3-1)*Phi_3[ID_2_3];
			else
				START_3[ID_2_3]=loopUpperLimits_3[ID_2_3]+(ID_1_3-1)*Phi_3[ID_2_3];
		}
	for(_CUDA_TILE_3=0;_CUDA_TILE_3<_NUM_TILE_3;_CUDA_TILE_3++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_3<<<_BLOCKS_3,_THREADS_3>>>(_DEV_a_3, _SZ_a_3, _DEV_x_3, _SZ_x_3, START_3[0], MIN(START_3[0]+1, 9), _CUDA_TILE_3);
			hipDeviceSynchronize();
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_3);
	#endif

	// ---------Copying Kernel variable from device to host----------
	hipMemcpy(x, _DEV_x_3, sizeof(int)*_SZ_x_3, hipMemcpyDeviceToHost);

	// ---------Releasing the memory allocated to kernel variable----------
	hipFree(_DEV_x_3);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_3);
		pre_time_3 = (double) ((((&mid_start_3)->tv_sec * 1000000000) + (&mid_start_3)->tv_nsec) - (((&start_3)->tv_sec * 1000000000) + (&start_3)->tv_nsec)) / 1000000000;
		post_time_3 = (double) ((((&end_3)->tv_sec * 1000000000) + (&end_3)->tv_nsec) - (((&mid_end_3)->tv_sec * 1000000000) + (&mid_end_3)->tv_nsec)) / 1000000000;
		computeTime_3 = (double) ((((&mid_end_3)->tv_sec * 1000000000) + (&mid_end_3)->tv_nsec) - (((&mid_start_3)->tv_sec * 1000000000) + (&mid_start_3)->tv_nsec)) / 1000000000;
		runTime_3 = (double) ((((&end_3)->tv_sec * 1000000000) + (&end_3)->tv_nsec) - (((&start_3)->tv_sec * 1000000000) + (&start_3)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_3.x,_BLOCKS_3.x,data,pre_time_3,computeTime_3,post_time_3,runTime_3,_CUDA_TILE_3);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_3);
	#endif
	#endif
}

// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_1(int* a,int _SZ_a_1,int* a,int _SZ_a_1,int CUDA_L_i_1,int CUDA_U_i_1, int _CUDA_TILE_1)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_1 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i_1<=i)&&(i<=CUDA_U_i_1)){
		a[i+10]=a[i+2];
}}



// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_2(int* a,int _SZ_a_2,int* w,int _SZ_w_2,int CUDA_L_i_2,int CUDA_U_i_2, int _CUDA_TILE_2)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_2 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i_2<=i)&&(i<=CUDA_U_i_2)){
		w[1+i]=w[1+i+1]+3.14/10;
}}



// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_3(int* a,int _SZ_a_3,int* x,int _SZ_x_3,int CUDA_L_i_3,int CUDA_U_i_3, int _CUDA_TILE_3)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_3 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i_3<=i)&&(i<=CUDA_U_i_3)){
		x[1+i]=x[1+i+1]+10;
}}

