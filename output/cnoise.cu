#include "hip/hip_runtime.h"
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_0(int* ,int ,int* ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
int main()
{
    int x[20];
    int w[20],i,j,k;
    for(i=0;i<20;i++)
    {
        x[i]=2*i;
        w[i]=2*i;
    }

   // PARALLELIZE IN CUDA START
    for( i=0; i<N; i++ ){
		x[i] = 2.0*fw[2*i];
	};
    // PARALLELIZE IN CUDA END
    
    return 0;
}


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start, end, mid_start, mid_end;
		double runTime, pre_time, post_time, computeTime;
	#endif
	int _SZ_w_0 = 20;
	int _SZ_x_0 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_w;
	hipMalloc((void**) &_DEV_w_0, sizeof(int)*_SZ_w_0);
	hipMemcpy(_DEV_w_0, w, sizeof(int)*_SZ_w_0, hipMemcpyHostToDevice);
	int *_DEV_x;
	hipMalloc((void**) &_DEV_x_0, sizeof(int)*_SZ_x_0);
	hipMemcpy(_DEV_x_0, x, sizeof(int)*_SZ_x_0, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_0 = 20;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start);
	#endif

	int _CUDA_TILE_0;
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>(_DEV_w_0, _SZ_w_0, _DEV_x_0, _SZ_x_0, 0, 0, _CUDA_TILE_0);
		hipDeviceSynchronize();
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end);
	#endif

	// ---------Copying Kernel variable from device to host----------

	// ---------Releasing the memory allocated to kernel variable----------

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end);
		pre_time = (double) ((((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
		post_time = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec)) / 1000000000;
		computeTime = (double) ((((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec) - (((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec)) / 1000000000;
		runTime = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time,computeTime,post_time,runTime,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%d\n",runTime);
	#endif
	#endif
}

// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_0(int* w,int _SZ_w_0,int* x,int _SZ_x_0,int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE_0)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_0 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
		x[i]=2.0*fw[2*i];
}}

