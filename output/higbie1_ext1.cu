#define _NTHREAD 512
#define _NBLOCK 65535

__global__ void _AFFINE_KERNEL(int* ,int ,int* ,int ,int ,int ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
int main()
{
    int XP1[20][20],XS3[20],i,j,k;
    for(i=0;i<20;i++)
    for(j=0;j<20;j++)
    {
        XP1[i][j]=i+j;
        XS3[i]=2*i;
    }
	int _SZ_XS3_1 = 20;
	int _SZ_XP1_2 = 20;
	int _SZ_XP1_1 = 20;
	int *_DEV_XS3;
	hipMalloc((void**) &_DEV_XS3, sizeof(int)*_SZ_XS3_1);
	hipMemcpy(_DEV_XS3, XS3, sizeof(int)*_SZ_XS3_1, hipMemcpyHostToDevice);
	int *_DEV_XP1;
	hipMalloc((void**) &_DEV_XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1);
	hipMemcpy(_DEV_XP1, XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1, hipMemcpyHostToDevice);
	float _NUM_THREADS = 400,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=20;
		_THREADS.y=20;
	}
	else {
		_NUM_BLOCKS=_NUM_THREADS/256;
		_BLOCKS.x=_BLOCKS.y=ceil(sqrt(_NUM_BLOCKS));
		_THREADS.x=_THREADS.y=ceil(sqrt(400.0/(_BLOCKS.x*_BLOCKS.y)));
		int temp=_NUM_BLOCKS;
		if(_NUM_BLOCKS>_NBLOCK)
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
	}
	int ID_1, ID_2, START[2];
	int _CUDA_TILE;
	int Phi[2]={3, 4};
	int loopUpperLimits[2]={20, 20};
	for(ID_1=1;ID_1<=MIN(20/3, 20/4)+1;ID_1++)
	{
		for(ID_2=0;ID_2<2;ID_2++)
		{
			if(Phi[ID_2]>=0)
				START[ID_2]=(ID_1-1)*Phi[ID_2];
			else
				START[ID_2]=loopUpperLimits[ID_2]+(ID_1-1)*Phi[ID_2];
		}
		for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		{
			_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_XS3, _SZ_XS3_1, _DEV_XP1, _SZ_XP1_2, _SZ_XP1_1, START[0], MIN(START[0]+3, 20), START[1], 20, _CUDA_TILE);
			hipDeviceSynchronize();
		}
		for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		{
			_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_XS3, _SZ_XS3_1, _DEV_XP1, _SZ_XP1_2, _SZ_XP1_1, START[0]+3, 20, START[1], MIN(START[1]+4, 20), _CUDA_TILE);
			hipDeviceSynchronize();
		}
	}
	hipMemcpy(XP1, _DEV_XP1, sizeof(int)*_SZ_XP1_2*_SZ_XP1_1, hipMemcpyDeviceToHost);
    return 0;
}



__global__ void _AFFINE_KERNEL(int* XS3,int _SZ_XS3_1,int* XP1,int _SZ_XP1_2,int _SZ_XP1_1,int CUDA_L_i,int CUDA_U_i, int CUDA_L_j,int CUDA_U_j, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	int j = gridDim.y*blockDim.y*_CUDA_TILE + blockDim.y*blockIdx.y + threadIdx.y;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
	if((CUDA_L_j<=j)&&(j<=CUDA_U_j)){
		XP1[i*_SZ_XP1_1+j]=XP1[(i+3)*_SZ_XP1_1+j+4]+XS3[i];
}}}

