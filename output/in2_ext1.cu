#include "hip/hip_runtime.h"
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_0(char ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
int main(int argc, char** argv)
{
    int N = 1000;
	int N_NODES = 100;
	int data = 1;
	//int _NTHREAD = 1, _NBLOCK = 1;
	char* readfile, *outfile;
	if(argc>1) _NTHREAD = atoi(argv[1]);
	if(argc>2) _NBLOCK = atoi(argv[2]);
	if(argc>3) data = atoi(argv[3]) + 1;
	if(argc>4) readfile = argv[4];
	int i,j;
	//srand(time(NULL));
	FILE* f;
	f = fopen(readfile, "r");
	j=0;
	char c;
	while(1){
		c = fgetc(f);
		if(c=='\n') { 
			j++;
			c = fgetc(f);
			if(c!='%') break;
		}
	}
	fscanf(f, "%d", &N_NODES);
	//printf("---------%d  ----------",N_NODES);
	fscanf(f, "%d", &N_NODES);
	fscanf(f, "%d", &N);
	if(2*N<_NTHREAD*_NBLOCK) {
		printf("%d\n",_NTHREAD*_NBLOCK);
		fclose(f);
		return 0;
	}
	struct timespec start, end, mid_start, mid_end;
	double runTime, pre_time, post_time, computeTime;
    int a[20],i,j,k;
    int n=20;
    for(i=0;i<20;i++)
    {
        a[i]=2*i;
    }
    clock_gettime(CLOCK_MONOTONIC, &start);
   // PARALLELIZE IN CUDA START
    for(i=18;i>0;i--)
    {
       a[i+1]=a[i-1];
       //y[2*i-3][3*j-5]=x[i][j];
       //x[3*i-2][2*j-4]=y[i][j];
     } 
    // PARALLELIZE IN CUDA END
    clock_gettime(CLOCK_MONOTONIC, &end);
	pre_time = (double) ((((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	post_time = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec)) / 1000000000;
	computeTime = (double) ((((&mid_end)->tv_sec * 1000000000) + (&mid_end)->tv_nsec) - (((&mid_start)->tv_sec * 1000000000) + (&mid_start)->tv_nsec)) / 1000000000;
	
	runTime = (double) ((((&end)->tv_sec * 1000000000) + (&end)->tv_nsec) - (((&start)->tv_sec * 1000000000) + (&start)->tv_nsec)) / 1000000000;
	printf("********************************\n");
	fprintf(fp,"%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,data,pre_time,computeTime,post_time,runTime,_CUDA_TILE);
	//fprintf(fp,"%d,%d,%.14f\n",N_EDGES,data,runTime);
	//fclose(fp);
	printf("RUN TIME: %.14f\n", runTime);

	fclose(fp);
	fclose(f);
    return 0;
}


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_a_0 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_a_0;
	hipMalloc((void**) &_DEV_a_0, sizeof(int)*_SZ_a_0);
	hipMemcpy(_DEV_a_0, a, sizeof(int)*_SZ_a_0, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_0 = 20;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int ID_1_0, ID_2_0, START_0[1];
	int _CUDA_TILE_0;
	int Phi_0[1]={2};
	int loopUpperLimits_0[1]={18};
	for(ID_1_0=1;ID_1_0<=18/2+1;ID_1_0++)
	{
		for(ID_2_0=0;ID_2_0<1;ID_2_0++)
		{
			if(Phi_0[ID_2_0]>=0)
				START_0[ID_2_0]=(ID_1_0-1)*Phi_0[ID_2_0];
			else
				START_0[ID_2_0]=loopUpperLimits_0[ID_2_0]+(ID_1_0-1)*Phi_0[ID_2_0];
		}
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>(c_0, _DEV_a_0, _SZ_a_0, START_0[0], MIN(START_0[0]+2, 18), _CUDA_TILE_0);
			hipDeviceSynchronize();
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif

	// ---------Copying Kernel variable from device to host----------
	hipMemcpy(a, _DEV_a_0, sizeof(int)*_SZ_a_0, hipMemcpyDeviceToHost);

	// ---------Releasing the memory allocated to kernel variable----------
	hipFree(_DEV_a_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);
		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_0);
	#endif
	#endif
}

// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_0(char c,int* a,int _SZ_a_0,int CUDA_L_i_0,int CUDA_U_i_0, int _CUDA_TILE_0)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_0 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i_0<=i)&&(i<=CUDA_U_i_0)){
		a[18-i+1]=a[18-i-1];
}}

