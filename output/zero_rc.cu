#include "hip/hip_runtime.h"
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_0(char ,int* ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
int main(int argc, char** argv)
{
    int N = 1000;
	int N_NODES = 100;
	int data = 1;
	//int _NTHREAD = 1, _NBLOCK = 1;
	char* readfile, *outfile;
	if(argc>1) _NTHREAD = atoi(argv[1]);
	if(argc>2) _NBLOCK = atoi(argv[2]);
	if(argc>3) data = atoi(argv[3]) + 1;
	if(argc>4) readfile = argv[4];
	int i,j;
	//srand(time(NULL));
	FILE* f;
	f = fopen(readfile, "r");
	j=0;
	char c;
	while(1){
		c = fgetc(f);
		if(c=='\n') { 
			j++;
			c = fgetc(f);
			if(c!='%') break;
		}
	}
	fscanf(f, "%d", &N_NODES);
	//printf("---------%d  ----------",N_NODES);
	fscanf(f, "%d", &N_NODES);
	fscanf(f, "%d", &N);
	if(2*N<_NTHREAD*_NBLOCK) {
		printf("%d\n",_NTHREAD*_NBLOCK);
		fclose(f);
		return 0;
	}
	struct timespec start, end, mid_start, mid_end;
	double runTime, pre_time, post_time, computeTime;
    int q[20],a[20];
    // PARALLELIZE IN CUDA START
    for(i=1;i<=10;i++)
    {
        q[i]=q[i+10]+10;
    }
     // PARALLELIZE IN CUDA END
 
	
    return 0;
}


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_q_0 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_q;
	hipMalloc((void**) &_DEV_q_0, sizeof(int)*_SZ_q_0);
	hipMemcpy(_DEV_q_0, q, sizeof(int)*_SZ_q_0, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_0 = 20;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int _CUDA_TILE_0;
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>(c_0, _DEV_q_0, _SZ_q_0, 0, 9, _CUDA_TILE_0);
		hipDeviceSynchronize();
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif

	// ---------Copying Kernel variable from device to host----------
	hipMemcpy(q, _DEV_q_0, sizeof(int)*_SZ_q_0, hipMemcpyDeviceToHost);

	// ---------Releasing the memory allocated to kernel variable----------
	hipFree(_DEV_q_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);
		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_0);
	#endif
	#endif
}

// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_0(char c,int* q,int _SZ_q_0,int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE_0)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_0 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
		q[1+i]=q[1+i+10]+10;
}}

