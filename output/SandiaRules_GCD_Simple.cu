#define _NTHREAD 512
#define _NBLOCK 65535
#include<hip/hip_runtime.h>

__global__ void _AFFINE_KERNEL(int* ,int ,int* ,int ,int ,int ,int ,int ,int );

#include<stdio.h>
#include<stdlib.h>
int main()
{
    int x[20];
    int w[20],i,j,k;
    for(i=0;i<20;i++)
    {
        x[i]=2*i;
        w[i]=2*i;
    }
	int _SZ_w_1 = 20;
	int _SZ_x_1 = 20;
	int *_DEV_w;
	hipMalloc((void**) &_DEV_w, sizeof(int)*_SZ_w_1);
	hipMemcpy(_DEV_w, w, sizeof(int)*_SZ_w_1, hipMemcpyHostToDevice);
	int *_DEV_x;
	hipMalloc((void**) &_DEV_x, sizeof(int)*_SZ_x_1);
	hipMemcpy(_DEV_x, x, sizeof(int)*_SZ_x_1, hipMemcpyHostToDevice);
	float _NUM_THREADS = 20,_NUM_BLOCKS=1;
	int _NUM_TILE=1;
	dim3 _THREADS(512);
	dim3 _BLOCKS(1);
	if(_NUM_THREADS < _NTHREAD)
	{
		_THREADS.x=_NUM_THREADS;
	}
	else {
		 _THREADS.x=_NTHREAD;
		_NUM_BLOCKS=_NUM_THREADS/512;
		if(_NUM_BLOCKS<_NBLOCK)
			_BLOCKS.x=_NUM_BLOCKS;
		else {
			_BLOCKS.x=_NBLOCK;
			int temp=_NUM_BLOCKS;
			_NUM_TILE=(temp % _NBLOCK == 0)?(_NUM_BLOCKS/_NBLOCK):((_NUM_BLOCKS/_NBLOCK)+1);
		}
	}
	int _CUDA_TILE;
	for(i=0;i<=9;i+=19)
	for(_CUDA_TILE=0;_CUDA_TILE<_NUM_TILE;_CUDA_TILE++)
		_AFFINE_KERNEL<<<_BLOCKS,_THREADS>>>(_DEV_w, _SZ_w_1, _DEV_x, _SZ_x_1, 1, i, 0, 9, _CUDA_TILE);
	hipDeviceSynchronize();
	hipMemcpy(w, _DEV_w, sizeof(int)*_SZ_w_1, hipMemcpyDeviceToHost);
	hipMemcpy(x, _DEV_x, sizeof(int)*_SZ_x_1, hipMemcpyDeviceToHost);
    
    return 0;
}



__global__ void _AFFINE_KERNEL(int* w,int _SZ_w_1,int* x,int _SZ_x_1,int phi_count, int CUDA_i, int CUDA_L_i,int CUDA_U_i, int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_i<=i)&&(i<(CUDA_i+19))&&(i<=CUDA_U_i)){
		x[1+i-1]=-x[20-1-i];
		w[1+i-1]=w[20-1-i];
}}

