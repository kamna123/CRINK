#include "hip/hip_runtime.h"
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_0(int* ,int ,int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>


// ----------KERNEL DECLARATION----------
__global__ void _AFFINE_KERNEL_2(int* ,int ,int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
int main()
{
    int x[10000],y[10000],i,a[10000];
    for(i=0;i<7900;i++)
    {
        x[i]=i;
        y[i]=2*i;
        a[i]=3*i;
    }
  
    for(i=5;i<=1900;i++)
    {
       x[i]=y[i-5];
       y[i]=x[i-4];
       //y[2*i-3][3*j-5]=x[i][j];
       //x[3*i-2][2*j-4]=y[i][j];
     } 
   
   // PARALLELIZE IN CUDA START
    for(i=0;i<8000;i++)
    {
     y[i+10]=a[i+4];
      x[i+4]=y[i+2];
       
     } 
     // PARALLELIZE IN CUDA END
     // PARALLELIZE IN CUDA START
    for(i=0;i<6000;i++)
    {
     y[a[i+2]]=y[a[i+4]]*1;
      x[i+2]=y[i+4];
       
     } 
     // PARALLELIZE IN CUDA END
  // PARALLELIZE IN CUDA START
    for(i=0;i<9000;i++)
    {
     y[i+2]=a[i+4];
      x[i+2]=y[i+4];
       
     } 
     // PARALLELIZE IN CUDA END
     
    
   
    return 0;
}


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_x_0 = 10000;
	int _SZ_a_0 = 10000;
	int _SZ_y_0 = 10000;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_x_0;
	hipMalloc((void**) &_DEV_x_0, sizeof(int)*_SZ_x_0);
	hipMemcpy(_DEV_x_0, x, sizeof(int)*_SZ_x_0, hipMemcpyHostToDevice);
	int *_DEV_a_0;
	hipMalloc((void**) &_DEV_a_0, sizeof(int)*_SZ_a_0);
	hipMemcpy(_DEV_a_0, a, sizeof(int)*_SZ_a_0, hipMemcpyHostToDevice);
	int *_DEV_y_0;
	hipMalloc((void**) &_DEV_y_0, sizeof(int)*_SZ_y_0);
	hipMemcpy(_DEV_y_0, y, sizeof(int)*_SZ_y_0, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_0 = 10000;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int ID_1_0, ID_2_0, START_0[1];
	int _CUDA_TILE_0;
	int Phi_0[1]={2};
	int loopUpperLimits_0[1]={8000};
	for(ID_1_0=1;ID_1_0<=8000/2+1;ID_1_0++)
	{
		for(ID_2_0=0;ID_2_0<1;ID_2_0++)
		{
			if(Phi_0[ID_2_0]>=0)
				START_0[ID_2_0]=(ID_1_0-1)*Phi_0[ID_2_0];
			else
				START_0[ID_2_0]=loopUpperLimits_0[ID_2_0]+(ID_1_0-1)*Phi_0[ID_2_0];
		}
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>(_DEV_x_0, _SZ_x_0, _DEV_a_0, _SZ_a_0, _DEV_y_0, _SZ_y_0, START_0[0], MIN(START_0[0]+2, 8000), _CUDA_TILE_0);
			hipDeviceSynchronize();
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif

	// ---------Copying Kernel variable from device to host----------
	hipMemcpy(y, _DEV_y_0, sizeof(int)*_SZ_y_0, hipMemcpyDeviceToHost);

	// ---------Releasing the memory allocated to kernel variable----------
	hipFree(_DEV_y_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);
		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_0);
	#endif
	#endif

	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_2, end_2, mid_start_2, mid_end_2;
		double runTime_2, pre_time_2, post_time_2, computeTime_2;
	#endif
	int _SZ_x_2 = 10000;
	int _SZ_a_2 = 10000;
	int _SZ_y_2 = 10000;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_2);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_x_2;
	hipMalloc((void**) &_DEV_x_2, sizeof(int)*_SZ_x_2);
	hipMemcpy(_DEV_x_2, x, sizeof(int)*_SZ_x_2, hipMemcpyHostToDevice);
	int *_DEV_a_2;
	hipMalloc((void**) &_DEV_a_2, sizeof(int)*_SZ_a_2);
	hipMemcpy(_DEV_a_2, a, sizeof(int)*_SZ_a_2, hipMemcpyHostToDevice);
	int *_DEV_y_2;
	hipMalloc((void**) &_DEV_y_2, sizeof(int)*_SZ_y_2);
	hipMemcpy(_DEV_y_2, y, sizeof(int)*_SZ_y_2, hipMemcpyHostToDevice);

	// ----------Tiling and declaring threads and blocks required for Kernel Execution----------
	int _NUM_THREADS_2 = 10000;
	float _NUM_BLOCKS_2=1;
	int _NUM_TILE_2=1;
	dim3 _THREADS_2(512);
	dim3 _BLOCKS_2(1);
	if(_NUM_THREADS_2 < _NTHREAD)
	{
		_THREADS_2.x=_NUM_THREADS_2;
	}
	else {
		 _THREADS_2.x=_NTHREAD;
		_NUM_BLOCKS_2=(_NUM_THREADS_2 % _NTHREAD == 0)?(_NUM_THREADS_2/_NTHREAD):((_NUM_THREADS_2/_NTHREAD)+1);
		if(_NUM_BLOCKS_2<_NBLOCK)
			_BLOCKS_2.x=_NUM_BLOCKS_2;
		else {
			_BLOCKS_2.x=_NBLOCK;
			int temp_2=_NUM_BLOCKS_2;
			_NUM_TILE_2=(temp_2 % _NBLOCK == 0)?(_NUM_BLOCKS_2/_NBLOCK):((_NUM_BLOCKS_2/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_2);
	#endif

	int ID_1_2, ID_2_2, START_2[1];
	int _CUDA_TILE_2;
	int Phi_2[1]={2};
	int loopUpperLimits_2[1]={9000};
	for(ID_1_2=1;ID_1_2<=9000/2+1;ID_1_2++)
	{
		for(ID_2_2=0;ID_2_2<1;ID_2_2++)
		{
			if(Phi_2[ID_2_2]>=0)
				START_2[ID_2_2]=(ID_1_2-1)*Phi_2[ID_2_2];
			else
				START_2[ID_2_2]=loopUpperLimits_2[ID_2_2]+(ID_1_2-1)*Phi_2[ID_2_2];
		}
	for(_CUDA_TILE_2=0;_CUDA_TILE_2<_NUM_TILE_2;_CUDA_TILE_2++)
	{
		// ----------KERNEL LAUNCH----------
		_AFFINE_KERNEL_2<<<_BLOCKS_2,_THREADS_2>>>(_DEV_x_2, _SZ_x_2, _DEV_a_2, _SZ_a_2, _DEV_y_2, _SZ_y_2, START_2[0], MIN(START_2[0]+2, 9000), _CUDA_TILE_2);
			hipDeviceSynchronize();
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_2);
	#endif

	// ---------Copying Kernel variable from device to host----------
	hipMemcpy(y, _DEV_y_2, sizeof(int)*_SZ_y_2, hipMemcpyDeviceToHost);

	// ---------Releasing the memory allocated to kernel variable----------
	hipFree(_DEV_y_2);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_2);
		pre_time_2 = (double) ((((&mid_start_2)->tv_sec * 1000000000) + (&mid_start_2)->tv_nsec) - (((&start_2)->tv_sec * 1000000000) + (&start_2)->tv_nsec)) / 1000000000;
		post_time_2 = (double) ((((&end_2)->tv_sec * 1000000000) + (&end_2)->tv_nsec) - (((&mid_end_2)->tv_sec * 1000000000) + (&mid_end_2)->tv_nsec)) / 1000000000;
		computeTime_2 = (double) ((((&mid_end_2)->tv_sec * 1000000000) + (&mid_end_2)->tv_nsec) - (((&mid_start_2)->tv_sec * 1000000000) + (&mid_start_2)->tv_nsec)) / 1000000000;
		runTime_2 = (double) ((((&end_2)->tv_sec * 1000000000) + (&end_2)->tv_nsec) - (((&start_2)->tv_sec * 1000000000) + (&start_2)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%d,%d,%d,%d,%d,%.14f,%.14f,%.14f,%.14f,%d\n",N,_NTHREAD*_NBLOCK,_THREADS_2.x,_BLOCKS_2.x,data,pre_time_2,computeTime_2,post_time_2,runTime_2,_CUDA_TILE_2);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%f\n",runTime_2);
	#endif
	#endif
}

// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_0(int* x,int _SZ_x_0,int* a,int _SZ_a_0,int* y,int _SZ_y_0,int CUDA_L_i_0,int CUDA_U_i_0, int _CUDA_TILE_0)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_0 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i_0<=i)&&(i<=CUDA_U_i_0)){
		y[i+10]=a[i+4];
		x[i+4]=y[i+2];
}}



// ----------KERNEL DEFINITION----------


__global__ void _AFFINE_KERNEL_2(int* x,int _SZ_x_2,int* a,int _SZ_a_2,int* y,int _SZ_y_2,int CUDA_L_i_2,int CUDA_U_i_2, int _CUDA_TILE_2)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE_2 + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i_2<=i)&&(i<=CUDA_U_i_2)){
		y[i+2]=a[i+4];
		x[i+2]=y[i+4];
}}

